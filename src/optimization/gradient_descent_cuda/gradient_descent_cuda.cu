#include "hip/hip_runtime.h"
#line 1 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"
//
// Created by Kai on 23.0f2.2022.
//

#include "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.h"

#include <random>
#include <iterator>
#include <algorithm>
#include <chrono>
#include <numeric>

#include <geometry/motor_estimation.h>
#include <optimization/fast_shuffle.h>

// CUDA
#include <cmath>
#include <Eigen/Core>

/*
 * CUDA HELPER FUNCTIONS ===============================================================================================
 */
inline unsigned int div_up(unsigned int numerator, unsigned int denominator)
{
    unsigned int result = numerator / denominator;
    if (numerator % denominator) ++result;
    return result;
}

dim3 getBlockDim() {
    // Choose a fixed (sensible) block size
    int block_size = 256;

    dim3 dimBlock(block_size, 1);
    return dimBlock;
}

dim3 getGridDim(int width, int height, dim3 blockSize) {
    dim3 dimGrid(div_up(width, blockSize.x), div_up(height, blockSize.y));

    return dimGrid;
}

#define CUDA_CHECK_ERROR                                                       \
    do {                                                                       \
        const hipError_t err = hipGetLastError();                            \
        if (err != hipSuccess) {                                              \
            const char *const err_str = hipGetErrorString(err);               \
            std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 1   \
                      << ": " << err_str << " (" << err << ")" << std::endl;   \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while(0)
/*
 * CUDA HELPER FUNCTIONS ===============================================================================================
 */

// Define the motor currently used for transforming as constant memory
__constant__ float constantMotor[8];

__device__ unsigned int xorshift32(unsigned int state) {
    state ^= state << 13;
    state ^= state >> 17;
    state ^= state << 5;
    return state;
}

/*
 * Takes a list of correspondences and indices and saves the result into outMotors
 */
__global__ void calculateMotors(const float* corr, const unsigned int* indices, float* outMotors, const int iteration, const int trianglesCount, const int correspondenceCount) {
    // 1D Thread ID
    int idx = threadIdx.x;

    // Load the correspondences into shared memory
    extern __shared__ float sharedCorrespondences[];

    // Load using all threads
    for (int i = idx; i < 6*correspondenceCount; i += blockDim.x) {
        sharedCorrespondences[i] = corr[i];
    }

    // If this thread is out of bounds for the triangle calculation: Skip
    if(idx > trianglesCount - 1) {
        return;
    }

    __syncthreads();

    // Generate three indices
    unsigned int id1 = 6*(xorshift32(133*idx) % correspondenceCount);
    unsigned int id2 = 6*(xorshift32(133*idx+1) % correspondenceCount);
    unsigned int id3 = 6*(xorshift32(133*idx+2) % correspondenceCount);

    // Load the current triangle into temp arrays
    float A_src_arr[4] = {sharedCorrespondences[id1], sharedCorrespondences[id1+1], sharedCorrespondences[id1+2], 1.0f};
    float B_src_arr[4] = {sharedCorrespondences[id2], sharedCorrespondences[id2+1], sharedCorrespondences[id2+2], 1.0f};
    float C_src_arr[4] = {sharedCorrespondences[id3], sharedCorrespondences[id3+1], sharedCorrespondences[id3+2], 1.0f};
    float A_tar_arr[4] = {sharedCorrespondences[id1+3], sharedCorrespondences[id1+4], sharedCorrespondences[id1+5], 1.0f};
    float B_tar_arr[4] = {sharedCorrespondences[id2+3], sharedCorrespondences[id2+4], sharedCorrespondences[id2+5], 1.0f};
    float C_tar_arr[4] = {sharedCorrespondences[id3+3], sharedCorrespondences[id3+4], sharedCorrespondences[id3+5], 1.0f};

    // Init the output
    float calculatedMotor[8] = {0.0};



#line 119 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"
#include <math.h>
//#pragma gpc multivector A_src
float A_src[8];
//#pragma gpc multivector A_src_raw
float A_src_raw[2];
//#pragma gpc multivector A_tar
float A_tar[2];
//#pragma gpc multivector B2
float B2[8];
//#pragma gpc multivector B_src
float B_src[8];
//#pragma gpc multivector B_src_raw
float B_src_raw[2];
//#pragma gpc multivector B_tar
float B_tar[2];
//#pragma gpc multivector C2
float C2[8];
//#pragma gpc multivector C3
float C3[4];
//#pragma gpc multivector C_src
float C_src[8];
//#pragma gpc multivector C_src_raw
float C_src_raw[2];
//#pragma gpc multivector C_tar
float C_tar[2];
//#pragma gpc multivector combined_motor
float combined_motor[8];
//#pragma gpc multivector L1
float L1[6];
//#pragma gpc multivector L2
float L2[7];
//#pragma gpc multivector motor_norm
float motor_norm;
//#pragma gpc multivector out_motor
float out_motor[8];
//#pragma gpc multivector P1
float P1[4];
//#pragma gpc multivector P2
float P2[4];
//#pragma gpc multivector VA
float VA[8];
//#pragma gpc multivector VA_norm
float VA_norm;
//#pragma gpc multivector VA_unnormalized
float VA_unnormalized[8];
//#pragma gpc multivector VB
float VB[8];
//#pragma gpc multivector VB_norm
float VB_norm;
//#pragma gpc multivector VB_unnormalized
float VB_unnormalized[8];
//#pragma gpc multivector VC
float VC[7];
//#pragma gpc multivector VC_norm
float VC_norm;
//#pragma gpc multivector VC_unnormalized
float VC_unnormalized[7];

//#pragma gpc multivector_component A_src_raw e0^e1^e2 A_src_raw[0]
A_src_raw[0] = (-A_src_arr[2]);
//#pragma gpc multivector_component A_src_raw e0^e2^e3 A_src_raw[1]
A_src_raw[1] = (-A_src_arr[0]);
//#pragma gpc multivector_component B_src_raw e0^e1^e2 B_src_raw[0]
B_src_raw[0] = (-B_src_arr[2]);
//#pragma gpc multivector_component B_src_raw e0^e2^e3 B_src_raw[1]
B_src_raw[1] = (-B_src_arr[0]);
//#pragma gpc multivector_component C_src_raw e0^e1^e2 C_src_raw[0]
C_src_raw[0] = (-C_src_arr[2]);
//#pragma gpc multivector_component C_src_raw e0^e2^e3 C_src_raw[1]
C_src_raw[1] = (-C_src_arr[0]);
//#pragma gpc multivector_component A_tar e0^e1^e2 A_tar[0]
A_tar[0] = (-A_tar_arr[2]);
//#pragma gpc multivector_component A_tar e0^e2^e3 A_tar[1]
A_tar[1] = (-A_tar_arr[0]);
//#pragma gpc multivector_component B_tar e0^e1^e2 B_tar[0]
B_tar[0] = (-B_tar_arr[2]);
//#pragma gpc multivector_component B_tar e0^e2^e3 B_tar[1]
B_tar[1] = (-B_tar_arr[0]);
//#pragma gpc multivector_component C_tar e0^e1^e2 C_tar[0]
C_tar[0] = (-C_tar_arr[2]);
//#pragma gpc multivector_component C_tar e0^e2^e3 C_tar[1]
C_tar[1] = (-C_tar_arr[0]);
//#pragma gpc multivector_component A_src e0 A_src[0]
A_src[0] = ((-(constantMotor[4] * A_src_raw[0])) + (-(constantMotor[5] * A_src_arr[1])) + (-(constantMotor[6] * A_src_raw[1])) + (-(constantMotor[7] * A_src_arr[3]))) * constantMotor[0] + (-((-(constantMotor[6] * A_src_arr[3])) * (-constantMotor[1]))) + (-(constantMotor[5] * A_src_arr[3] * (-constantMotor[2]))) + (-((-(constantMotor[4] * A_src_arr[3])) * (-constantMotor[3]))) + (-((constantMotor[0] * A_src_raw[0] + constantMotor[3] * A_src_arr[3] + (-(constantMotor[5] * A_src_raw[1])) + constantMotor[6] * A_src_arr[1]) * (-constantMotor[4]))) + (-((constantMotor[0] * A_src_arr[1] + (-(constantMotor[2] * A_src_arr[3])) + constantMotor[4] * A_src_raw[1] + (-(constantMotor[6] * A_src_raw[0]))) * (-constantMotor[5]))) + (-((constantMotor[0] * A_src_raw[1] + constantMotor[1] * A_src_arr[3] + (-(constantMotor[4] * A_src_arr[1])) + constantMotor[5] * A_src_raw[0]) * (-constantMotor[6]))) + constantMotor[0] * A_src_arr[3] * constantMotor[7];
//#pragma gpc multivector_component A_src e1 A_src[1]
A_src[1] = (-(constantMotor[6] * A_src_arr[3])) * constantMotor[0] + (-(constantMotor[5] * A_src_arr[3] * (-constantMotor[4]))) + (-((-(constantMotor[4] * A_src_arr[3])) * (-constantMotor[5]))) + (-(constantMotor[0] * A_src_arr[3] * (-constantMotor[6])));
//#pragma gpc multivector_component A_src e2 A_src[2]
A_src[2] = (-(constantMotor[6] * A_src_arr[3])) * (-constantMotor[4]) + constantMotor[5] * A_src_arr[3] * constantMotor[0] + (-((-(constantMotor[4] * A_src_arr[3])) * (-constantMotor[6]))) + constantMotor[0] * A_src_arr[3] * (-constantMotor[5]);
//#pragma gpc multivector_component A_src e3 A_src[3]
A_src[3] = (-(constantMotor[6] * A_src_arr[3])) * (-constantMotor[5]) + constantMotor[5] * A_src_arr[3] * (-constantMotor[6]) + (-(constantMotor[4] * A_src_arr[3])) * constantMotor[0] + (-(constantMotor[0] * A_src_arr[3] * (-constantMotor[4])));
//#pragma gpc multivector_component A_src e0^e1^e2 A_src[4]
A_src[4] = ((-(constantMotor[4] * A_src_raw[0])) + (-(constantMotor[5] * A_src_arr[1])) + (-(constantMotor[6] * A_src_raw[1])) + (-(constantMotor[7] * A_src_arr[3]))) * (-constantMotor[4]) + (-((-(constantMotor[6] * A_src_arr[3])) * (-constantMotor[2]))) + constantMotor[5] * A_src_arr[3] * (-constantMotor[1]) + (-((-(constantMotor[4] * A_src_arr[3])) * constantMotor[7])) + (constantMotor[0] * A_src_raw[0] + constantMotor[3] * A_src_arr[3] + (-(constantMotor[5] * A_src_raw[1])) + constantMotor[6] * A_src_arr[1]) * constantMotor[0] + (-((constantMotor[0] * A_src_arr[1] + (-(constantMotor[2] * A_src_arr[3])) + constantMotor[4] * A_src_raw[1] + (-(constantMotor[6] * A_src_raw[0]))) * (-constantMotor[6]))) + (constantMotor[0] * A_src_raw[1] + constantMotor[1] * A_src_arr[3] + (-(constantMotor[4] * A_src_arr[1])) + constantMotor[5] * A_src_raw[0]) * (-constantMotor[5]) + (-(constantMotor[0] * A_src_arr[3] * (-constantMotor[3])));
//#pragma gpc multivector_component A_src e0^e1^e3 A_src[5]
A_src[5] = ((-(constantMotor[4] * A_src_raw[0])) + (-(constantMotor[5] * A_src_arr[1])) + (-(constantMotor[6] * A_src_raw[1])) + (-(constantMotor[7] * A_src_arr[3]))) * (-constantMotor[5]) + (-((-(constantMotor[6] * A_src_arr[3])) * (-constantMotor[3]))) + constantMotor[5] * A_src_arr[3] * constantMotor[7] + (-(constantMotor[4] * A_src_arr[3])) * (-constantMotor[1]) + (constantMotor[0] * A_src_raw[0] + constantMotor[3] * A_src_arr[3] + (-(constantMotor[5] * A_src_raw[1])) + constantMotor[6] * A_src_arr[1]) * (-constantMotor[6]) + (constantMotor[0] * A_src_arr[1] + (-(constantMotor[2] * A_src_arr[3])) + constantMotor[4] * A_src_raw[1] + (-(constantMotor[6] * A_src_raw[0]))) * constantMotor[0] + (-((constantMotor[0] * A_src_raw[1] + constantMotor[1] * A_src_arr[3] + (-(constantMotor[4] * A_src_arr[1])) + constantMotor[5] * A_src_raw[0]) * (-constantMotor[4]))) + constantMotor[0] * A_src_arr[3] * (-constantMotor[2]);
//#pragma gpc multivector_component A_src e0^e2^e3 A_src[6]
A_src[6] = ((-(constantMotor[4] * A_src_raw[0])) + (-(constantMotor[5] * A_src_arr[1])) + (-(constantMotor[6] * A_src_raw[1])) + (-(constantMotor[7] * A_src_arr[3]))) * (-constantMotor[6]) + (-((-(constantMotor[6] * A_src_arr[3])) * constantMotor[7])) + (-(constantMotor[5] * A_src_arr[3] * (-constantMotor[3]))) + (-(constantMotor[4] * A_src_arr[3])) * (-constantMotor[2]) + (-((constantMotor[0] * A_src_raw[0] + constantMotor[3] * A_src_arr[3] + (-(constantMotor[5] * A_src_raw[1])) + constantMotor[6] * A_src_arr[1]) * (-constantMotor[5]))) + (constantMotor[0] * A_src_arr[1] + (-(constantMotor[2] * A_src_arr[3])) + constantMotor[4] * A_src_raw[1] + (-(constantMotor[6] * A_src_raw[0]))) * (-constantMotor[4]) + (constantMotor[0] * A_src_raw[1] + constantMotor[1] * A_src_arr[3] + (-(constantMotor[4] * A_src_arr[1])) + constantMotor[5] * A_src_raw[0]) * constantMotor[0] + (-(constantMotor[0] * A_src_arr[3] * (-constantMotor[1])));
//#pragma gpc multivector_component A_src e1^e2^e3 A_src[7]
A_src[7] = (-(constantMotor[6] * A_src_arr[3])) * (-constantMotor[6]) + (-(constantMotor[5] * A_src_arr[3] * (-constantMotor[5]))) + (-(constantMotor[4] * A_src_arr[3])) * (-constantMotor[4]) + constantMotor[0] * A_src_arr[3] * constantMotor[0];
//#pragma gpc multivector_component B_src e0 B_src[0]
B_src[0] = ((-(constantMotor[4] * B_src_raw[0])) + (-(constantMotor[5] * B_src_arr[1])) + (-(constantMotor[6] * B_src_raw[1])) + (-(constantMotor[7] * B_src_arr[3]))) * constantMotor[0] + (-((-(constantMotor[6] * B_src_arr[3])) * (-constantMotor[1]))) + (-(constantMotor[5] * B_src_arr[3] * (-constantMotor[2]))) + (-((-(constantMotor[4] * B_src_arr[3])) * (-constantMotor[3]))) + (-((constantMotor[0] * B_src_raw[0] + constantMotor[3] * B_src_arr[3] + (-(constantMotor[5] * B_src_raw[1])) + constantMotor[6] * B_src_arr[1]) * (-constantMotor[4]))) + (-((constantMotor[0] * B_src_arr[1] + (-(constantMotor[2] * B_src_arr[3])) + constantMotor[4] * B_src_raw[1] + (-(constantMotor[6] * B_src_raw[0]))) * (-constantMotor[5]))) + (-((constantMotor[0] * B_src_raw[1] + constantMotor[1] * B_src_arr[3] + (-(constantMotor[4] * B_src_arr[1])) + constantMotor[5] * B_src_raw[0]) * (-constantMotor[6]))) + constantMotor[0] * B_src_arr[3] * constantMotor[7];
//#pragma gpc multivector_component B_src e1 B_src[1]
B_src[1] = (-(constantMotor[6] * B_src_arr[3])) * constantMotor[0] + (-(constantMotor[5] * B_src_arr[3] * (-constantMotor[4]))) + (-((-(constantMotor[4] * B_src_arr[3])) * (-constantMotor[5]))) + (-(constantMotor[0] * B_src_arr[3] * (-constantMotor[6])));
//#pragma gpc multivector_component B_src e2 B_src[2]
B_src[2] = (-(constantMotor[6] * B_src_arr[3])) * (-constantMotor[4]) + constantMotor[5] * B_src_arr[3] * constantMotor[0] + (-((-(constantMotor[4] * B_src_arr[3])) * (-constantMotor[6]))) + constantMotor[0] * B_src_arr[3] * (-constantMotor[5]);
//#pragma gpc multivector_component B_src e3 B_src[3]
B_src[3] = (-(constantMotor[6] * B_src_arr[3])) * (-constantMotor[5]) + constantMotor[5] * B_src_arr[3] * (-constantMotor[6]) + (-(constantMotor[4] * B_src_arr[3])) * constantMotor[0] + (-(constantMotor[0] * B_src_arr[3] * (-constantMotor[4])));
//#pragma gpc multivector_component B_src e0^e1^e2 B_src[4]
B_src[4] = ((-(constantMotor[4] * B_src_raw[0])) + (-(constantMotor[5] * B_src_arr[1])) + (-(constantMotor[6] * B_src_raw[1])) + (-(constantMotor[7] * B_src_arr[3]))) * (-constantMotor[4]) + (-((-(constantMotor[6] * B_src_arr[3])) * (-constantMotor[2]))) + constantMotor[5] * B_src_arr[3] * (-constantMotor[1]) + (-((-(constantMotor[4] * B_src_arr[3])) * constantMotor[7])) + (constantMotor[0] * B_src_raw[0] + constantMotor[3] * B_src_arr[3] + (-(constantMotor[5] * B_src_raw[1])) + constantMotor[6] * B_src_arr[1]) * constantMotor[0] + (-((constantMotor[0] * B_src_arr[1] + (-(constantMotor[2] * B_src_arr[3])) + constantMotor[4] * B_src_raw[1] + (-(constantMotor[6] * B_src_raw[0]))) * (-constantMotor[6]))) + (constantMotor[0] * B_src_raw[1] + constantMotor[1] * B_src_arr[3] + (-(constantMotor[4] * B_src_arr[1])) + constantMotor[5] * B_src_raw[0]) * (-constantMotor[5]) + (-(constantMotor[0] * B_src_arr[3] * (-constantMotor[3])));
//#pragma gpc multivector_component B_src e0^e1^e3 B_src[5]
B_src[5] = ((-(constantMotor[4] * B_src_raw[0])) + (-(constantMotor[5] * B_src_arr[1])) + (-(constantMotor[6] * B_src_raw[1])) + (-(constantMotor[7] * B_src_arr[3]))) * (-constantMotor[5]) + (-((-(constantMotor[6] * B_src_arr[3])) * (-constantMotor[3]))) + constantMotor[5] * B_src_arr[3] * constantMotor[7] + (-(constantMotor[4] * B_src_arr[3])) * (-constantMotor[1]) + (constantMotor[0] * B_src_raw[0] + constantMotor[3] * B_src_arr[3] + (-(constantMotor[5] * B_src_raw[1])) + constantMotor[6] * B_src_arr[1]) * (-constantMotor[6]) + (constantMotor[0] * B_src_arr[1] + (-(constantMotor[2] * B_src_arr[3])) + constantMotor[4] * B_src_raw[1] + (-(constantMotor[6] * B_src_raw[0]))) * constantMotor[0] + (-((constantMotor[0] * B_src_raw[1] + constantMotor[1] * B_src_arr[3] + (-(constantMotor[4] * B_src_arr[1])) + constantMotor[5] * B_src_raw[0]) * (-constantMotor[4]))) + constantMotor[0] * B_src_arr[3] * (-constantMotor[2]);
//#pragma gpc multivector_component B_src e0^e2^e3 B_src[6]
B_src[6] = ((-(constantMotor[4] * B_src_raw[0])) + (-(constantMotor[5] * B_src_arr[1])) + (-(constantMotor[6] * B_src_raw[1])) + (-(constantMotor[7] * B_src_arr[3]))) * (-constantMotor[6]) + (-((-(constantMotor[6] * B_src_arr[3])) * constantMotor[7])) + (-(constantMotor[5] * B_src_arr[3] * (-constantMotor[3]))) + (-(constantMotor[4] * B_src_arr[3])) * (-constantMotor[2]) + (-((constantMotor[0] * B_src_raw[0] + constantMotor[3] * B_src_arr[3] + (-(constantMotor[5] * B_src_raw[1])) + constantMotor[6] * B_src_arr[1]) * (-constantMotor[5]))) + (constantMotor[0] * B_src_arr[1] + (-(constantMotor[2] * B_src_arr[3])) + constantMotor[4] * B_src_raw[1] + (-(constantMotor[6] * B_src_raw[0]))) * (-constantMotor[4]) + (constantMotor[0] * B_src_raw[1] + constantMotor[1] * B_src_arr[3] + (-(constantMotor[4] * B_src_arr[1])) + constantMotor[5] * B_src_raw[0]) * constantMotor[0] + (-(constantMotor[0] * B_src_arr[3] * (-constantMotor[1])));
//#pragma gpc multivector_component B_src e1^e2^e3 B_src[7]
B_src[7] = (-(constantMotor[6] * B_src_arr[3])) * (-constantMotor[6]) + (-(constantMotor[5] * B_src_arr[3] * (-constantMotor[5]))) + (-(constantMotor[4] * B_src_arr[3])) * (-constantMotor[4]) + constantMotor[0] * B_src_arr[3] * constantMotor[0];
//#pragma gpc multivector_component C_src e0 C_src[0]
C_src[0] = ((-(constantMotor[4] * C_src_raw[0])) + (-(constantMotor[5] * C_src_arr[1])) + (-(constantMotor[6] * C_src_raw[1])) + (-(constantMotor[7] * C_src_arr[3]))) * constantMotor[0] + (-((-(constantMotor[6] * C_src_arr[3])) * (-constantMotor[1]))) + (-(constantMotor[5] * C_src_arr[3] * (-constantMotor[2]))) + (-((-(constantMotor[4] * C_src_arr[3])) * (-constantMotor[3]))) + (-((constantMotor[0] * C_src_raw[0] + constantMotor[3] * C_src_arr[3] + (-(constantMotor[5] * C_src_raw[1])) + constantMotor[6] * C_src_arr[1]) * (-constantMotor[4]))) + (-((constantMotor[0] * C_src_arr[1] + (-(constantMotor[2] * C_src_arr[3])) + constantMotor[4] * C_src_raw[1] + (-(constantMotor[6] * C_src_raw[0]))) * (-constantMotor[5]))) + (-((constantMotor[0] * C_src_raw[1] + constantMotor[1] * C_src_arr[3] + (-(constantMotor[4] * C_src_arr[1])) + constantMotor[5] * C_src_raw[0]) * (-constantMotor[6]))) + constantMotor[0] * C_src_arr[3] * constantMotor[7];
//#pragma gpc multivector_component C_src e1 C_src[1]
C_src[1] = (-(constantMotor[6] * C_src_arr[3])) * constantMotor[0] + (-(constantMotor[5] * C_src_arr[3] * (-constantMotor[4]))) + (-((-(constantMotor[4] * C_src_arr[3])) * (-constantMotor[5]))) + (-(constantMotor[0] * C_src_arr[3] * (-constantMotor[6])));
//#pragma gpc multivector_component C_src e2 C_src[2]
C_src[2] = (-(constantMotor[6] * C_src_arr[3])) * (-constantMotor[4]) + constantMotor[5] * C_src_arr[3] * constantMotor[0] + (-((-(constantMotor[4] * C_src_arr[3])) * (-constantMotor[6]))) + constantMotor[0] * C_src_arr[3] * (-constantMotor[5]);
//#pragma gpc multivector_component C_src e3 C_src[3]
C_src[3] = (-(constantMotor[6] * C_src_arr[3])) * (-constantMotor[5]) + constantMotor[5] * C_src_arr[3] * (-constantMotor[6]) + (-(constantMotor[4] * C_src_arr[3])) * constantMotor[0] + (-(constantMotor[0] * C_src_arr[3] * (-constantMotor[4])));
//#pragma gpc multivector_component C_src e0^e1^e2 C_src[4]
C_src[4] = ((-(constantMotor[4] * C_src_raw[0])) + (-(constantMotor[5] * C_src_arr[1])) + (-(constantMotor[6] * C_src_raw[1])) + (-(constantMotor[7] * C_src_arr[3]))) * (-constantMotor[4]) + (-((-(constantMotor[6] * C_src_arr[3])) * (-constantMotor[2]))) + constantMotor[5] * C_src_arr[3] * (-constantMotor[1]) + (-((-(constantMotor[4] * C_src_arr[3])) * constantMotor[7])) + (constantMotor[0] * C_src_raw[0] + constantMotor[3] * C_src_arr[3] + (-(constantMotor[5] * C_src_raw[1])) + constantMotor[6] * C_src_arr[1]) * constantMotor[0] + (-((constantMotor[0] * C_src_arr[1] + (-(constantMotor[2] * C_src_arr[3])) + constantMotor[4] * C_src_raw[1] + (-(constantMotor[6] * C_src_raw[0]))) * (-constantMotor[6]))) + (constantMotor[0] * C_src_raw[1] + constantMotor[1] * C_src_arr[3] + (-(constantMotor[4] * C_src_arr[1])) + constantMotor[5] * C_src_raw[0]) * (-constantMotor[5]) + (-(constantMotor[0] * C_src_arr[3] * (-constantMotor[3])));
//#pragma gpc multivector_component C_src e0^e1^e3 C_src[5]
C_src[5] = ((-(constantMotor[4] * C_src_raw[0])) + (-(constantMotor[5] * C_src_arr[1])) + (-(constantMotor[6] * C_src_raw[1])) + (-(constantMotor[7] * C_src_arr[3]))) * (-constantMotor[5]) + (-((-(constantMotor[6] * C_src_arr[3])) * (-constantMotor[3]))) + constantMotor[5] * C_src_arr[3] * constantMotor[7] + (-(constantMotor[4] * C_src_arr[3])) * (-constantMotor[1]) + (constantMotor[0] * C_src_raw[0] + constantMotor[3] * C_src_arr[3] + (-(constantMotor[5] * C_src_raw[1])) + constantMotor[6] * C_src_arr[1]) * (-constantMotor[6]) + (constantMotor[0] * C_src_arr[1] + (-(constantMotor[2] * C_src_arr[3])) + constantMotor[4] * C_src_raw[1] + (-(constantMotor[6] * C_src_raw[0]))) * constantMotor[0] + (-((constantMotor[0] * C_src_raw[1] + constantMotor[1] * C_src_arr[3] + (-(constantMotor[4] * C_src_arr[1])) + constantMotor[5] * C_src_raw[0]) * (-constantMotor[4]))) + constantMotor[0] * C_src_arr[3] * (-constantMotor[2]);
//#pragma gpc multivector_component C_src e0^e2^e3 C_src[6]
C_src[6] = ((-(constantMotor[4] * C_src_raw[0])) + (-(constantMotor[5] * C_src_arr[1])) + (-(constantMotor[6] * C_src_raw[1])) + (-(constantMotor[7] * C_src_arr[3]))) * (-constantMotor[6]) + (-((-(constantMotor[6] * C_src_arr[3])) * constantMotor[7])) + (-(constantMotor[5] * C_src_arr[3] * (-constantMotor[3]))) + (-(constantMotor[4] * C_src_arr[3])) * (-constantMotor[2]) + (-((constantMotor[0] * C_src_raw[0] + constantMotor[3] * C_src_arr[3] + (-(constantMotor[5] * C_src_raw[1])) + constantMotor[6] * C_src_arr[1]) * (-constantMotor[5]))) + (constantMotor[0] * C_src_arr[1] + (-(constantMotor[2] * C_src_arr[3])) + constantMotor[4] * C_src_raw[1] + (-(constantMotor[6] * C_src_raw[0]))) * (-constantMotor[4]) + (constantMotor[0] * C_src_raw[1] + constantMotor[1] * C_src_arr[3] + (-(constantMotor[4] * C_src_arr[1])) + constantMotor[5] * C_src_raw[0]) * constantMotor[0] + (-(constantMotor[0] * C_src_arr[3] * (-constantMotor[1])));
//#pragma gpc multivector_component C_src e1^e2^e3 C_src[7]
C_src[7] = (-(constantMotor[6] * C_src_arr[3])) * (-constantMotor[6]) + (-(constantMotor[5] * C_src_arr[3] * (-constantMotor[5]))) + (-(constantMotor[4] * C_src_arr[3])) * (-constantMotor[4]) + constantMotor[0] * C_src_arr[3] * constantMotor[0];
//#pragma gpc multivector_component VA_unnormalized 1.0 VA_unnormalized[0]
VA_unnormalized[0] = 1.0 + (-(A_tar_arr[3] * (-A_src[7]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))))));
//#pragma gpc multivector_component VA_unnormalized e0^e1 VA_unnormalized[1]
VA_unnormalized[1] = A_tar[0] * A_src[2] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))) + A_tar_arr[1] * A_src[3] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))) + (-(A_tar[1] * (-A_src[7]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))))) + A_tar_arr[3] * (-A_src[6]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))));
//#pragma gpc multivector_component VA_unnormalized e0^e2 VA_unnormalized[2]
VA_unnormalized[2] = (-(A_tar[0] * A_src[1] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))))) + A_tar_arr[1] * (-A_src[7]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))) + A_tar[1] * A_src[3] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))) + (-(A_tar_arr[3] * (-A_src[5]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))))));
//#pragma gpc multivector_component VA_unnormalized e0^e3 VA_unnormalized[3]
VA_unnormalized[3] = (-(A_tar[0] * (-A_src[7]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))))) + (-(A_tar_arr[1] * A_src[1] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))))) + (-(A_tar[1] * A_src[2] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))))) + A_tar_arr[3] * (-A_src[4]) / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))));
//#pragma gpc multivector_component VA_unnormalized e1^e2 VA_unnormalized[4]
VA_unnormalized[4] = A_tar_arr[3] * A_src[3] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))));
//#pragma gpc multivector_component VA_unnormalized e1^e3 VA_unnormalized[5]
VA_unnormalized[5] = (-(A_tar_arr[3] * A_src[2] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))))));
//#pragma gpc multivector_component VA_unnormalized e2^e3 VA_unnormalized[6]
VA_unnormalized[6] = A_tar_arr[3] * A_src[1] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))));
//#pragma gpc multivector_component VA_unnormalized e0^e1^e2^e3 VA_unnormalized[7]
VA_unnormalized[7] = A_tar[0] * A_src[3] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))) + (-(A_tar_arr[1] * A_src[2] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))))) + A_tar[1] * A_src[1] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7])))) + (-(A_tar_arr[3] * A_src[0] / (A_src[1] * A_src[1] + A_src[2] * A_src[2] + A_src[3] * A_src[3] + (-(A_src[7] * (-A_src[7]))))));
//#pragma gpc multivector_component VA_norm 1.0 VA_norm
VA_norm = sqrtf(fabs(VA_unnormalized[0] * VA_unnormalized[0] + (-(VA_unnormalized[4] * (-VA_unnormalized[4]))) + (-(VA_unnormalized[5] * (-VA_unnormalized[5]))) + (-(VA_unnormalized[6] * (-VA_unnormalized[6])))));
//#pragma gpc multivector_component VA 1.0 VA[0]
VA[0] = VA_unnormalized[0] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e0^e1 VA[1]
VA[1] = VA_unnormalized[1] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e0^e2 VA[2]
VA[2] = VA_unnormalized[2] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e0^e3 VA[3]
VA[3] = VA_unnormalized[3] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e1^e2 VA[4]
VA[4] = VA_unnormalized[4] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e1^e3 VA[5]
VA[5] = VA_unnormalized[5] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e2^e3 VA[6]
VA[6] = VA_unnormalized[6] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component VA e0^e1^e2^e3 VA[7]
VA[7] = VA_unnormalized[7] * VA_norm / (VA_norm * VA_norm);
//#pragma gpc multivector_component B2 e0 B2[0]
B2[0] = (VA[0] * B_src[0] + VA[1] * B_src[1] + VA[2] * B_src[2] + VA[3] * B_src[3] + (-(VA[4] * B_src[4])) + (-(VA[5] * B_src[5])) + (-(VA[6] * B_src[6])) + (-(VA[7] * B_src[7]))) * VA[0] + (-((VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * (-VA[1]))) + (-((VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * (-VA[2]))) + (-((VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * (-VA[3]))) + (-((VA[0] * B_src[4] + VA[1] * B_src[2] + (-(VA[2] * B_src[1])) + VA[3] * B_src[7] + VA[4] * B_src[0] + (-(VA[5] * B_src[6])) + VA[6] * B_src[5] + VA[7] * B_src[3]) * (-VA[4]))) + (-((VA[0] * B_src[5] + VA[1] * B_src[3] + (-(VA[2] * B_src[7])) + (-(VA[3] * B_src[1])) + VA[4] * B_src[6] + VA[5] * B_src[0] + (-(VA[6] * B_src[4])) + (-(VA[7] * B_src[2]))) * (-VA[5]))) + (-((VA[0] * B_src[6] + VA[1] * B_src[7] + VA[2] * B_src[3] + (-(VA[3] * B_src[2])) + (-(VA[4] * B_src[5])) + VA[5] * B_src[4] + VA[6] * B_src[0] + VA[7] * B_src[1]) * (-VA[6]))) + (VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * VA[7];
//#pragma gpc multivector_component B2 e1 B2[1]
B2[1] = (VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * VA[0] + (-((VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * (-VA[4]))) + (-((VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * (-VA[5]))) + (-((VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * (-VA[6])));
//#pragma gpc multivector_component B2 e2 B2[2]
B2[2] = (VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * (-VA[4]) + (VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * VA[0] + (-((VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * (-VA[6]))) + (VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * (-VA[5]);
//#pragma gpc multivector_component B2 e3 B2[3]
B2[3] = (VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * (-VA[5]) + (VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * (-VA[6]) + (VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * VA[0] + (-((VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * (-VA[4])));
//#pragma gpc multivector_component B2 e0^e1^e2 B2[4]
B2[4] = (VA[0] * B_src[0] + VA[1] * B_src[1] + VA[2] * B_src[2] + VA[3] * B_src[3] + (-(VA[4] * B_src[4])) + (-(VA[5] * B_src[5])) + (-(VA[6] * B_src[6])) + (-(VA[7] * B_src[7]))) * (-VA[4]) + (-((VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * (-VA[2]))) + (VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * (-VA[1]) + (-((VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * VA[7])) + (VA[0] * B_src[4] + VA[1] * B_src[2] + (-(VA[2] * B_src[1])) + VA[3] * B_src[7] + VA[4] * B_src[0] + (-(VA[5] * B_src[6])) + VA[6] * B_src[5] + VA[7] * B_src[3]) * VA[0] + (-((VA[0] * B_src[5] + VA[1] * B_src[3] + (-(VA[2] * B_src[7])) + (-(VA[3] * B_src[1])) + VA[4] * B_src[6] + VA[5] * B_src[0] + (-(VA[6] * B_src[4])) + (-(VA[7] * B_src[2]))) * (-VA[6]))) + (VA[0] * B_src[6] + VA[1] * B_src[7] + VA[2] * B_src[3] + (-(VA[3] * B_src[2])) + (-(VA[4] * B_src[5])) + VA[5] * B_src[4] + VA[6] * B_src[0] + VA[7] * B_src[1]) * (-VA[5]) + (-((VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * (-VA[3])));
//#pragma gpc multivector_component B2 e0^e1^e3 B2[5]
B2[5] = (VA[0] * B_src[0] + VA[1] * B_src[1] + VA[2] * B_src[2] + VA[3] * B_src[3] + (-(VA[4] * B_src[4])) + (-(VA[5] * B_src[5])) + (-(VA[6] * B_src[6])) + (-(VA[7] * B_src[7]))) * (-VA[5]) + (-((VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * (-VA[3]))) + (VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * VA[7] + (VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * (-VA[1]) + (VA[0] * B_src[4] + VA[1] * B_src[2] + (-(VA[2] * B_src[1])) + VA[3] * B_src[7] + VA[4] * B_src[0] + (-(VA[5] * B_src[6])) + VA[6] * B_src[5] + VA[7] * B_src[3]) * (-VA[6]) + (VA[0] * B_src[5] + VA[1] * B_src[3] + (-(VA[2] * B_src[7])) + (-(VA[3] * B_src[1])) + VA[4] * B_src[6] + VA[5] * B_src[0] + (-(VA[6] * B_src[4])) + (-(VA[7] * B_src[2]))) * VA[0] + (-((VA[0] * B_src[6] + VA[1] * B_src[7] + VA[2] * B_src[3] + (-(VA[3] * B_src[2])) + (-(VA[4] * B_src[5])) + VA[5] * B_src[4] + VA[6] * B_src[0] + VA[7] * B_src[1]) * (-VA[4]))) + (VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * (-VA[2]);
//#pragma gpc multivector_component B2 e0^e2^e3 B2[6]
B2[6] = (VA[0] * B_src[0] + VA[1] * B_src[1] + VA[2] * B_src[2] + VA[3] * B_src[3] + (-(VA[4] * B_src[4])) + (-(VA[5] * B_src[5])) + (-(VA[6] * B_src[6])) + (-(VA[7] * B_src[7]))) * (-VA[6]) + (-((VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * VA[7])) + (-((VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * (-VA[3]))) + (VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * (-VA[2]) + (-((VA[0] * B_src[4] + VA[1] * B_src[2] + (-(VA[2] * B_src[1])) + VA[3] * B_src[7] + VA[4] * B_src[0] + (-(VA[5] * B_src[6])) + VA[6] * B_src[5] + VA[7] * B_src[3]) * (-VA[5]))) + (VA[0] * B_src[5] + VA[1] * B_src[3] + (-(VA[2] * B_src[7])) + (-(VA[3] * B_src[1])) + VA[4] * B_src[6] + VA[5] * B_src[0] + (-(VA[6] * B_src[4])) + (-(VA[7] * B_src[2]))) * (-VA[4]) + (VA[0] * B_src[6] + VA[1] * B_src[7] + VA[2] * B_src[3] + (-(VA[3] * B_src[2])) + (-(VA[4] * B_src[5])) + VA[5] * B_src[4] + VA[6] * B_src[0] + VA[7] * B_src[1]) * VA[0] + (-((VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * (-VA[1])));
//#pragma gpc multivector_component B2 e1^e2^e3 B2[7]
B2[7] = (VA[0] * B_src[1] + VA[4] * B_src[2] + VA[5] * B_src[3] + (-(VA[6] * B_src[7]))) * (-VA[6]) + (-((VA[0] * B_src[2] + (-(VA[4] * B_src[1])) + VA[5] * B_src[7] + VA[6] * B_src[3]) * (-VA[5]))) + (VA[0] * B_src[3] + (-(VA[4] * B_src[7])) + (-(VA[5] * B_src[1])) + (-(VA[6] * B_src[2]))) * (-VA[4]) + (VA[0] * B_src[7] + VA[4] * B_src[3] + (-(VA[5] * B_src[2])) + VA[6] * B_src[1]) * VA[0];
//#pragma gpc multivector_component C2 e0 C2[0]
C2[0] = (VA[0] * C_src[0] + VA[1] * C_src[1] + VA[2] * C_src[2] + VA[3] * C_src[3] + (-(VA[4] * C_src[4])) + (-(VA[5] * C_src[5])) + (-(VA[6] * C_src[6])) + (-(VA[7] * C_src[7]))) * VA[0] + (-((VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * (-VA[1]))) + (-((VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * (-VA[2]))) + (-((VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * (-VA[3]))) + (-((VA[0] * C_src[4] + VA[1] * C_src[2] + (-(VA[2] * C_src[1])) + VA[3] * C_src[7] + VA[4] * C_src[0] + (-(VA[5] * C_src[6])) + VA[6] * C_src[5] + VA[7] * C_src[3]) * (-VA[4]))) + (-((VA[0] * C_src[5] + VA[1] * C_src[3] + (-(VA[2] * C_src[7])) + (-(VA[3] * C_src[1])) + VA[4] * C_src[6] + VA[5] * C_src[0] + (-(VA[6] * C_src[4])) + (-(VA[7] * C_src[2]))) * (-VA[5]))) + (-((VA[0] * C_src[6] + VA[1] * C_src[7] + VA[2] * C_src[3] + (-(VA[3] * C_src[2])) + (-(VA[4] * C_src[5])) + VA[5] * C_src[4] + VA[6] * C_src[0] + VA[7] * C_src[1]) * (-VA[6]))) + (VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * VA[7];
//#pragma gpc multivector_component C2 e1 C2[1]
C2[1] = (VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * VA[0] + (-((VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * (-VA[4]))) + (-((VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * (-VA[5]))) + (-((VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * (-VA[6])));
//#pragma gpc multivector_component C2 e2 C2[2]
C2[2] = (VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * (-VA[4]) + (VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * VA[0] + (-((VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * (-VA[6]))) + (VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * (-VA[5]);
//#pragma gpc multivector_component C2 e3 C2[3]
C2[3] = (VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * (-VA[5]) + (VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * (-VA[6]) + (VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * VA[0] + (-((VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * (-VA[4])));
//#pragma gpc multivector_component C2 e0^e1^e2 C2[4]
C2[4] = (VA[0] * C_src[0] + VA[1] * C_src[1] + VA[2] * C_src[2] + VA[3] * C_src[3] + (-(VA[4] * C_src[4])) + (-(VA[5] * C_src[5])) + (-(VA[6] * C_src[6])) + (-(VA[7] * C_src[7]))) * (-VA[4]) + (-((VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * (-VA[2]))) + (VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * (-VA[1]) + (-((VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * VA[7])) + (VA[0] * C_src[4] + VA[1] * C_src[2] + (-(VA[2] * C_src[1])) + VA[3] * C_src[7] + VA[4] * C_src[0] + (-(VA[5] * C_src[6])) + VA[6] * C_src[5] + VA[7] * C_src[3]) * VA[0] + (-((VA[0] * C_src[5] + VA[1] * C_src[3] + (-(VA[2] * C_src[7])) + (-(VA[3] * C_src[1])) + VA[4] * C_src[6] + VA[5] * C_src[0] + (-(VA[6] * C_src[4])) + (-(VA[7] * C_src[2]))) * (-VA[6]))) + (VA[0] * C_src[6] + VA[1] * C_src[7] + VA[2] * C_src[3] + (-(VA[3] * C_src[2])) + (-(VA[4] * C_src[5])) + VA[5] * C_src[4] + VA[6] * C_src[0] + VA[7] * C_src[1]) * (-VA[5]) + (-((VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * (-VA[3])));
//#pragma gpc multivector_component C2 e0^e1^e3 C2[5]
C2[5] = (VA[0] * C_src[0] + VA[1] * C_src[1] + VA[2] * C_src[2] + VA[3] * C_src[3] + (-(VA[4] * C_src[4])) + (-(VA[5] * C_src[5])) + (-(VA[6] * C_src[6])) + (-(VA[7] * C_src[7]))) * (-VA[5]) + (-((VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * (-VA[3]))) + (VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * VA[7] + (VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * (-VA[1]) + (VA[0] * C_src[4] + VA[1] * C_src[2] + (-(VA[2] * C_src[1])) + VA[3] * C_src[7] + VA[4] * C_src[0] + (-(VA[5] * C_src[6])) + VA[6] * C_src[5] + VA[7] * C_src[3]) * (-VA[6]) + (VA[0] * C_src[5] + VA[1] * C_src[3] + (-(VA[2] * C_src[7])) + (-(VA[3] * C_src[1])) + VA[4] * C_src[6] + VA[5] * C_src[0] + (-(VA[6] * C_src[4])) + (-(VA[7] * C_src[2]))) * VA[0] + (-((VA[0] * C_src[6] + VA[1] * C_src[7] + VA[2] * C_src[3] + (-(VA[3] * C_src[2])) + (-(VA[4] * C_src[5])) + VA[5] * C_src[4] + VA[6] * C_src[0] + VA[7] * C_src[1]) * (-VA[4]))) + (VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * (-VA[2]);
//#pragma gpc multivector_component C2 e0^e2^e3 C2[6]
C2[6] = (VA[0] * C_src[0] + VA[1] * C_src[1] + VA[2] * C_src[2] + VA[3] * C_src[3] + (-(VA[4] * C_src[4])) + (-(VA[5] * C_src[5])) + (-(VA[6] * C_src[6])) + (-(VA[7] * C_src[7]))) * (-VA[6]) + (-((VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * VA[7])) + (-((VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * (-VA[3]))) + (VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * (-VA[2]) + (-((VA[0] * C_src[4] + VA[1] * C_src[2] + (-(VA[2] * C_src[1])) + VA[3] * C_src[7] + VA[4] * C_src[0] + (-(VA[5] * C_src[6])) + VA[6] * C_src[5] + VA[7] * C_src[3]) * (-VA[5]))) + (VA[0] * C_src[5] + VA[1] * C_src[3] + (-(VA[2] * C_src[7])) + (-(VA[3] * C_src[1])) + VA[4] * C_src[6] + VA[5] * C_src[0] + (-(VA[6] * C_src[4])) + (-(VA[7] * C_src[2]))) * (-VA[4]) + (VA[0] * C_src[6] + VA[1] * C_src[7] + VA[2] * C_src[3] + (-(VA[3] * C_src[2])) + (-(VA[4] * C_src[5])) + VA[5] * C_src[4] + VA[6] * C_src[0] + VA[7] * C_src[1]) * VA[0] + (-((VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * (-VA[1])));
//#pragma gpc multivector_component C2 e1^e2^e3 C2[7]
C2[7] = (VA[0] * C_src[1] + VA[4] * C_src[2] + VA[5] * C_src[3] + (-(VA[6] * C_src[7]))) * (-VA[6]) + (-((VA[0] * C_src[2] + (-(VA[4] * C_src[1])) + VA[5] * C_src[7] + VA[6] * C_src[3]) * (-VA[5]))) + (VA[0] * C_src[3] + (-(VA[4] * C_src[7])) + (-(VA[5] * C_src[1])) + (-(VA[6] * C_src[2]))) * (-VA[4]) + (VA[0] * C_src[7] + VA[4] * C_src[3] + (-(VA[5] * C_src[2])) + VA[6] * C_src[1]) * VA[0];
//#pragma gpc multivector_component L1 e0^e1 L1[0]
L1[0] = A_tar_arr[1] * (-B_tar[0]) + (-((-A_tar[0]) * B_tar_arr[1]));
//#pragma gpc multivector_component L1 e0^e2 L1[1]
L1[1] = (-((-A_tar[1]) * (-B_tar[0]) + (-((-A_tar[0]) * (-B_tar[1])))));
//#pragma gpc multivector_component L1 e0^e3 L1[2]
L1[2] = (-A_tar[1]) * B_tar_arr[1] + (-(A_tar_arr[1] * (-B_tar[1])));
//#pragma gpc multivector_component L1 e1^e2 L1[3]
L1[3] = A_tar_arr[3] * (-B_tar[0]) + (-((-A_tar[0]) * B_tar_arr[3]));
//#pragma gpc multivector_component L1 e1^e3 L1[4]
L1[4] = (-(A_tar_arr[3] * B_tar_arr[1] + (-(A_tar_arr[1] * B_tar_arr[3]))));
//#pragma gpc multivector_component L1 e2^e3 L1[5]
L1[5] = A_tar_arr[3] * (-B_tar[1]) + (-((-A_tar[1]) * B_tar_arr[3]));
//#pragma gpc multivector_component L2 1.0 L2[0]
L2[0] = A_tar_arr[3] * B2[0] + (-((-A_tar[1]) * (-B2[1]))) + A_tar_arr[1] * B2[2] + (-((-A_tar[0]) * (-B2[3])));
//#pragma gpc multivector_component L2 e0^e1 L2[1]
L2[1] = A_tar_arr[1] * (-B2[4]) + (-((-A_tar[0]) * B2[5]));
//#pragma gpc multivector_component L2 e0^e2 L2[2]
L2[2] = (-((-A_tar[1]) * (-B2[4]) + (-((-A_tar[0]) * (-B2[6])))));
//#pragma gpc multivector_component L2 e0^e3 L2[3]
L2[3] = (-A_tar[1]) * B2[5] + (-(A_tar_arr[1] * (-B2[6])));
//#pragma gpc multivector_component L2 e1^e2 L2[4]
L2[4] = A_tar_arr[3] * (-B2[4]) + (-((-A_tar[0]) * B2[7]));
//#pragma gpc multivector_component L2 e1^e3 L2[5]
L2[5] = (-(A_tar_arr[3] * B2[5] + (-(A_tar_arr[1] * B2[7]))));
//#pragma gpc multivector_component L2 e2^e3 L2[6]
L2[6] = A_tar_arr[3] * (-B2[6]) + (-((-A_tar[1]) * B2[7]));
//#pragma gpc multivector_component VB_unnormalized 1.0 VB_unnormalized[0]
VB_unnormalized[0] = 1.0 + (-(L1[3] * (-L2[4]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + (-(L1[4] * (-L2[5]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + (-(L1[5] * (-L2[6]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))))));
//#pragma gpc multivector_component VB_unnormalized e0^e1 VB_unnormalized[1]
VB_unnormalized[1] = L1[0] * L2[0] / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[1] * (-L2[4]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + (-(L1[2] * (-L2[5]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + L1[3] * (-L2[2]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[4] * (-L2[3]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))));
//#pragma gpc multivector_component VB_unnormalized e0^e2 VB_unnormalized[2]
VB_unnormalized[2] = L1[0] * (-L2[4]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[1] * L2[0] / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[2] * (-L2[6]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + (-(L1[3] * (-L2[1]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + L1[5] * (-L2[3]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))));
//#pragma gpc multivector_component VB_unnormalized e0^e3 VB_unnormalized[3]
VB_unnormalized[3] = L1[0] * (-L2[5]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[1] * (-L2[6]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[2] * L2[0] / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[4] * (-L2[1]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + (-(L1[5] * (-L2[2]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))))));
//#pragma gpc multivector_component VB_unnormalized e1^e2 VB_unnormalized[4]
VB_unnormalized[4] = L1[3] * L2[0] / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[4] * (-L2[6]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + L1[5] * (-L2[5]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))));
//#pragma gpc multivector_component VB_unnormalized e1^e3 VB_unnormalized[5]
VB_unnormalized[5] = L1[3] * (-L2[6]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[4] * L2[0] / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[5] * (-L2[4]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))))));
//#pragma gpc multivector_component VB_unnormalized e2^e3 VB_unnormalized[6]
VB_unnormalized[6] = (-(L1[3] * (-L2[5]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + L1[4] * (-L2[4]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[5] * L2[0] / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))));
//#pragma gpc multivector_component VB_unnormalized e0^e1^e2^e3 VB_unnormalized[7]
VB_unnormalized[7] = L1[0] * (-L2[6]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[1] * (-L2[5]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + L1[2] * (-L2[4]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + L1[3] * (-L2[3]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))) + (-(L1[4] * (-L2[2]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6])))))) + L1[5] * (-L2[1]) / (L2[0] * L2[0] + (-(L2[4] * (-L2[4]))) + (-(L2[5] * (-L2[5]))) + (-(L2[6] * (-L2[6]))));
//#pragma gpc multivector_component VB_norm 1.0 VB_norm
VB_norm = sqrtf(fabs(VB_unnormalized[0] * VB_unnormalized[0] + (-(VB_unnormalized[4] * (-VB_unnormalized[4]))) + (-(VB_unnormalized[5] * (-VB_unnormalized[5]))) + (-(VB_unnormalized[6] * (-VB_unnormalized[6])))));
//#pragma gpc multivector_component VB 1.0 VB[0]
VB[0] = VB_unnormalized[0] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e0^e1 VB[1]
VB[1] = VB_unnormalized[1] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e0^e2 VB[2]
VB[2] = VB_unnormalized[2] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e0^e3 VB[3]
VB[3] = VB_unnormalized[3] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e1^e2 VB[4]
VB[4] = VB_unnormalized[4] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e1^e3 VB[5]
VB[5] = VB_unnormalized[5] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e2^e3 VB[6]
VB[6] = VB_unnormalized[6] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component VB e0^e1^e2^e3 VB[7]
VB[7] = VB_unnormalized[7] * VB_norm / (VB_norm * VB_norm);
//#pragma gpc multivector_component C3 e0^e1^e2 C3[0]
C3[0] = (VB[0] * C2[0] + VB[1] * C2[1] + VB[2] * C2[2] + VB[3] * C2[3] + (-(VB[4] * C2[4])) + (-(VB[5] * C2[5])) + (-(VB[6] * C2[6])) + (-(VB[7] * C2[7]))) * (-VB[4]) + (-((VB[0] * C2[1] + VB[4] * C2[2] + VB[5] * C2[3] + (-(VB[6] * C2[7]))) * (-VB[2]))) + (VB[0] * C2[2] + (-(VB[4] * C2[1])) + VB[5] * C2[7] + VB[6] * C2[3]) * (-VB[1]) + (-((VB[0] * C2[3] + (-(VB[4] * C2[7])) + (-(VB[5] * C2[1])) + (-(VB[6] * C2[2]))) * VB[7])) + (VB[0] * C2[4] + VB[1] * C2[2] + (-(VB[2] * C2[1])) + VB[3] * C2[7] + VB[4] * C2[0] + (-(VB[5] * C2[6])) + VB[6] * C2[5] + VB[7] * C2[3]) * VB[0] + (-((VB[0] * C2[5] + VB[1] * C2[3] + (-(VB[2] * C2[7])) + (-(VB[3] * C2[1])) + VB[4] * C2[6] + VB[5] * C2[0] + (-(VB[6] * C2[4])) + (-(VB[7] * C2[2]))) * (-VB[6]))) + (VB[0] * C2[6] + VB[1] * C2[7] + VB[2] * C2[3] + (-(VB[3] * C2[2])) + (-(VB[4] * C2[5])) + VB[5] * C2[4] + VB[6] * C2[0] + VB[7] * C2[1]) * (-VB[5]) + (-((VB[0] * C2[7] + VB[4] * C2[3] + (-(VB[5] * C2[2])) + VB[6] * C2[1]) * (-VB[3])));
//#pragma gpc multivector_component C3 e0^e1^e3 C3[1]
C3[1] = (VB[0] * C2[0] + VB[1] * C2[1] + VB[2] * C2[2] + VB[3] * C2[3] + (-(VB[4] * C2[4])) + (-(VB[5] * C2[5])) + (-(VB[6] * C2[6])) + (-(VB[7] * C2[7]))) * (-VB[5]) + (-((VB[0] * C2[1] + VB[4] * C2[2] + VB[5] * C2[3] + (-(VB[6] * C2[7]))) * (-VB[3]))) + (VB[0] * C2[2] + (-(VB[4] * C2[1])) + VB[5] * C2[7] + VB[6] * C2[3]) * VB[7] + (VB[0] * C2[3] + (-(VB[4] * C2[7])) + (-(VB[5] * C2[1])) + (-(VB[6] * C2[2]))) * (-VB[1]) + (VB[0] * C2[4] + VB[1] * C2[2] + (-(VB[2] * C2[1])) + VB[3] * C2[7] + VB[4] * C2[0] + (-(VB[5] * C2[6])) + VB[6] * C2[5] + VB[7] * C2[3]) * (-VB[6]) + (VB[0] * C2[5] + VB[1] * C2[3] + (-(VB[2] * C2[7])) + (-(VB[3] * C2[1])) + VB[4] * C2[6] + VB[5] * C2[0] + (-(VB[6] * C2[4])) + (-(VB[7] * C2[2]))) * VB[0] + (-((VB[0] * C2[6] + VB[1] * C2[7] + VB[2] * C2[3] + (-(VB[3] * C2[2])) + (-(VB[4] * C2[5])) + VB[5] * C2[4] + VB[6] * C2[0] + VB[7] * C2[1]) * (-VB[4]))) + (VB[0] * C2[7] + VB[4] * C2[3] + (-(VB[5] * C2[2])) + VB[6] * C2[1]) * (-VB[2]);
//#pragma gpc multivector_component C3 e0^e2^e3 C3[2]
C3[2] = (VB[0] * C2[0] + VB[1] * C2[1] + VB[2] * C2[2] + VB[3] * C2[3] + (-(VB[4] * C2[4])) + (-(VB[5] * C2[5])) + (-(VB[6] * C2[6])) + (-(VB[7] * C2[7]))) * (-VB[6]) + (-((VB[0] * C2[1] + VB[4] * C2[2] + VB[5] * C2[3] + (-(VB[6] * C2[7]))) * VB[7])) + (-((VB[0] * C2[2] + (-(VB[4] * C2[1])) + VB[5] * C2[7] + VB[6] * C2[3]) * (-VB[3]))) + (VB[0] * C2[3] + (-(VB[4] * C2[7])) + (-(VB[5] * C2[1])) + (-(VB[6] * C2[2]))) * (-VB[2]) + (-((VB[0] * C2[4] + VB[1] * C2[2] + (-(VB[2] * C2[1])) + VB[3] * C2[7] + VB[4] * C2[0] + (-(VB[5] * C2[6])) + VB[6] * C2[5] + VB[7] * C2[3]) * (-VB[5]))) + (VB[0] * C2[5] + VB[1] * C2[3] + (-(VB[2] * C2[7])) + (-(VB[3] * C2[1])) + VB[4] * C2[6] + VB[5] * C2[0] + (-(VB[6] * C2[4])) + (-(VB[7] * C2[2]))) * (-VB[4]) + (VB[0] * C2[6] + VB[1] * C2[7] + VB[2] * C2[3] + (-(VB[3] * C2[2])) + (-(VB[4] * C2[5])) + VB[5] * C2[4] + VB[6] * C2[0] + VB[7] * C2[1]) * VB[0] + (-((VB[0] * C2[7] + VB[4] * C2[3] + (-(VB[5] * C2[2])) + VB[6] * C2[1]) * (-VB[1])));
//#pragma gpc multivector_component C3 e1^e2^e3 C3[3]
C3[3] = (VB[0] * C2[1] + VB[4] * C2[2] + VB[5] * C2[3] + (-(VB[6] * C2[7]))) * (-VB[6]) + (-((VB[0] * C2[2] + (-(VB[4] * C2[1])) + VB[5] * C2[7] + VB[6] * C2[3]) * (-VB[5]))) + (VB[0] * C2[3] + (-(VB[4] * C2[7])) + (-(VB[5] * C2[1])) + (-(VB[6] * C2[2]))) * (-VB[4]) + (VB[0] * C2[7] + VB[4] * C2[3] + (-(VB[5] * C2[2])) + VB[6] * C2[1]) * VB[0];
//#pragma gpc multivector_component P1 e0 P1[0]
P1[0] = L1[2] * (-C_tar[0]) + (-((-L1[1]) * C_tar_arr[1])) + L1[0] * (-C_tar[1]);
//#pragma gpc multivector_component P1 e1 P1[1]
P1[1] = (-((-L1[4]) * (-C_tar[0]) + (-(L1[3] * C_tar_arr[1])) + L1[0] * C_tar_arr[3]));
//#pragma gpc multivector_component P1 e2 P1[2]
P1[2] = L1[5] * (-C_tar[0]) + (-(L1[3] * (-C_tar[1]))) + (-L1[1]) * C_tar_arr[3];
//#pragma gpc multivector_component P1 e3 P1[3]
P1[3] = (-(L1[5] * C_tar_arr[1] + (-((-L1[4]) * (-C_tar[1]))) + L1[2] * C_tar_arr[3]));
//#pragma gpc multivector_component P2 e0 P2[0]
P2[0] = L1[2] * (-C3[0]) + (-((-L1[1]) * C3[1])) + L1[0] * (-C3[2]);
//#pragma gpc multivector_component P2 e1 P2[1]
P2[1] = (-((-L1[4]) * (-C3[0]) + (-(L1[3] * C3[1])) + L1[0] * C3[3]));
//#pragma gpc multivector_component P2 e2 P2[2]
P2[2] = L1[5] * (-C3[0]) + (-(L1[3] * (-C3[2]))) + (-L1[1]) * C3[3];
//#pragma gpc multivector_component P2 e3 P2[3]
P2[3] = (-(L1[5] * C3[1] + (-((-L1[4]) * (-C3[2]))) + L1[2] * C3[3]));
//#pragma gpc multivector_component VC_unnormalized 1.0 VC_unnormalized[0]
VC_unnormalized[0] = 1.0 + P1[1] * P2[1] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + P1[2] * P2[2] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + P1[3] * P2[3] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]);
//#pragma gpc multivector_component VC_unnormalized e0^e1 VC_unnormalized[1]
VC_unnormalized[1] = P1[0] * P2[1] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + (-(P1[1] * P2[0] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3])));
//#pragma gpc multivector_component VC_unnormalized e0^e2 VC_unnormalized[2]
VC_unnormalized[2] = P1[0] * P2[2] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + (-(P1[2] * P2[0] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3])));
//#pragma gpc multivector_component VC_unnormalized e0^e3 VC_unnormalized[3]
VC_unnormalized[3] = P1[0] * P2[3] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + (-(P1[3] * P2[0] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3])));
//#pragma gpc multivector_component VC_unnormalized e1^e2 VC_unnormalized[4]
VC_unnormalized[4] = P1[1] * P2[2] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + (-(P1[2] * P2[1] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3])));
//#pragma gpc multivector_component VC_unnormalized e1^e3 VC_unnormalized[5]
VC_unnormalized[5] = P1[1] * P2[3] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + (-(P1[3] * P2[1] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3])));
//#pragma gpc multivector_component VC_unnormalized e2^e3 VC_unnormalized[6]
VC_unnormalized[6] = P1[2] * P2[3] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3]) + (-(P1[3] * P2[2] / (P2[1] * P2[1] + P2[2] * P2[2] + P2[3] * P2[3])));
//#pragma gpc multivector_component VC_norm 1.0 VC_norm
VC_norm = sqrtf(fabs(VC_unnormalized[0] * VC_unnormalized[0] + (-(VC_unnormalized[4] * (-VC_unnormalized[4]))) + (-(VC_unnormalized[5] * (-VC_unnormalized[5]))) + (-(VC_unnormalized[6] * (-VC_unnormalized[6])))));
//#pragma gpc multivector_component VC 1.0 VC[0]
VC[0] = VC_unnormalized[0] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component VC e0^e1 VC[1]
VC[1] = VC_unnormalized[1] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component VC e0^e2 VC[2]
VC[2] = VC_unnormalized[2] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component VC e0^e3 VC[3]
VC[3] = VC_unnormalized[3] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component VC e1^e2 VC[4]
VC[4] = VC_unnormalized[4] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component VC e1^e3 VC[5]
VC[5] = VC_unnormalized[5] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component VC e2^e3 VC[6]
VC[6] = VC_unnormalized[6] * VC_norm / (VC_norm * VC_norm);
//#pragma gpc multivector_component combined_motor 1.0 combined_motor[0]
combined_motor[0] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[0] + (-((VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[4])) + (-((VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[5])) + (-((VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[6]));
//#pragma gpc multivector_component combined_motor e0^e1 combined_motor[1]
combined_motor[1] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[1] + (VC[0] * VB[1] + VC[1] * VB[0] + (-(VC[2] * VB[4])) + (-(VC[3] * VB[5])) + VC[4] * VB[2] + VC[5] * VB[3] + (-(VC[6] * VB[7]))) * VA[0] + (-((VC[0] * VB[2] + VC[1] * VB[4] + VC[2] * VB[0] + (-(VC[3] * VB[6])) + (-(VC[4] * VB[1])) + VC[5] * VB[7] + VC[6] * VB[3]) * VA[4])) + (-((VC[0] * VB[3] + VC[1] * VB[5] + VC[2] * VB[6] + VC[3] * VB[0] + (-(VC[4] * VB[7])) + (-(VC[5] * VB[1])) + (-(VC[6] * VB[2]))) * VA[5])) + (VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[2] + (VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[3] + (-((VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[7])) + (-((VC[0] * VB[7] + VC[1] * VB[6] + (-(VC[2] * VB[5])) + VC[3] * VB[4] + VC[4] * VB[3] + (-(VC[5] * VB[2])) + VC[6] * VB[1]) * VA[6]));
//#pragma gpc multivector_component combined_motor e0^e2 combined_motor[2]
combined_motor[2] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[2] + (VC[0] * VB[1] + VC[1] * VB[0] + (-(VC[2] * VB[4])) + (-(VC[3] * VB[5])) + VC[4] * VB[2] + VC[5] * VB[3] + (-(VC[6] * VB[7]))) * VA[4] + (VC[0] * VB[2] + VC[1] * VB[4] + VC[2] * VB[0] + (-(VC[3] * VB[6])) + (-(VC[4] * VB[1])) + VC[5] * VB[7] + VC[6] * VB[3]) * VA[0] + (-((VC[0] * VB[3] + VC[1] * VB[5] + VC[2] * VB[6] + VC[3] * VB[0] + (-(VC[4] * VB[7])) + (-(VC[5] * VB[1])) + (-(VC[6] * VB[2]))) * VA[6])) + (-((VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[1])) + (VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[7] + (VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[3] + (VC[0] * VB[7] + VC[1] * VB[6] + (-(VC[2] * VB[5])) + VC[3] * VB[4] + VC[4] * VB[3] + (-(VC[5] * VB[2])) + VC[6] * VB[1]) * VA[5];
//#pragma gpc multivector_component combined_motor e0^e3 combined_motor[3]
combined_motor[3] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[3] + (VC[0] * VB[1] + VC[1] * VB[0] + (-(VC[2] * VB[4])) + (-(VC[3] * VB[5])) + VC[4] * VB[2] + VC[5] * VB[3] + (-(VC[6] * VB[7]))) * VA[5] + (VC[0] * VB[2] + VC[1] * VB[4] + VC[2] * VB[0] + (-(VC[3] * VB[6])) + (-(VC[4] * VB[1])) + VC[5] * VB[7] + VC[6] * VB[3]) * VA[6] + (VC[0] * VB[3] + VC[1] * VB[5] + VC[2] * VB[6] + VC[3] * VB[0] + (-(VC[4] * VB[7])) + (-(VC[5] * VB[1])) + (-(VC[6] * VB[2]))) * VA[0] + (-((VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[7])) + (-((VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[1])) + (-((VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[2])) + (-((VC[0] * VB[7] + VC[1] * VB[6] + (-(VC[2] * VB[5])) + VC[3] * VB[4] + VC[4] * VB[3] + (-(VC[5] * VB[2])) + VC[6] * VB[1]) * VA[4]));
//#pragma gpc multivector_component combined_motor e1^e2 combined_motor[4]
combined_motor[4] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[4] + (VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[0] + (-((VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[6])) + (VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[5];
//#pragma gpc multivector_component combined_motor e1^e3 combined_motor[5]
combined_motor[5] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[5] + (VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[6] + (VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[0] + (-((VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[4]));
//#pragma gpc multivector_component combined_motor e2^e3 combined_motor[6]
combined_motor[6] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[6] + (-((VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[5])) + (VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[4] + (VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[0];
//#pragma gpc multivector_component combined_motor e0^e1^e2^e3 combined_motor[7]
combined_motor[7] = (VC[0] * VB[0] + (-(VC[4] * VB[4])) + (-(VC[5] * VB[5])) + (-(VC[6] * VB[6]))) * VA[7] + (VC[0] * VB[1] + VC[1] * VB[0] + (-(VC[2] * VB[4])) + (-(VC[3] * VB[5])) + VC[4] * VB[2] + VC[5] * VB[3] + (-(VC[6] * VB[7]))) * VA[6] + (-((VC[0] * VB[2] + VC[1] * VB[4] + VC[2] * VB[0] + (-(VC[3] * VB[6])) + (-(VC[4] * VB[1])) + VC[5] * VB[7] + VC[6] * VB[3]) * VA[5])) + (VC[0] * VB[3] + VC[1] * VB[5] + VC[2] * VB[6] + VC[3] * VB[0] + (-(VC[4] * VB[7])) + (-(VC[5] * VB[1])) + (-(VC[6] * VB[2]))) * VA[4] + (VC[0] * VB[4] + VC[4] * VB[0] + (-(VC[5] * VB[6])) + VC[6] * VB[5]) * VA[3] + (-((VC[0] * VB[5] + VC[4] * VB[6] + VC[5] * VB[0] + (-(VC[6] * VB[4]))) * VA[2])) + (VC[0] * VB[6] + (-(VC[4] * VB[5])) + VC[5] * VB[4] + VC[6] * VB[0]) * VA[1] + (VC[0] * VB[7] + VC[1] * VB[6] + (-(VC[2] * VB[5])) + VC[3] * VB[4] + VC[4] * VB[3] + (-(VC[5] * VB[2])) + VC[6] * VB[1]) * VA[0];
//#pragma gpc multivector_component motor_norm 1.0 motor_norm
motor_norm = sqrtf(fabs(combined_motor[0] * combined_motor[0] + (-(combined_motor[4] * (-combined_motor[4]))) + (-(combined_motor[5] * (-combined_motor[5]))) + (-(combined_motor[6] * (-combined_motor[6])))));
//#pragma gpc multivector_component out_motor 1.0 out_motor[0]
out_motor[0] = combined_motor[0] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e0^e1 out_motor[1]
out_motor[1] = combined_motor[1] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e0^e2 out_motor[2]
out_motor[2] = combined_motor[2] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e0^e3 out_motor[3]
out_motor[3] = combined_motor[3] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e1^e2 out_motor[4]
out_motor[4] = combined_motor[4] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e1^e3 out_motor[5]
out_motor[5] = combined_motor[5] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e2^e3 out_motor[6]
out_motor[6] = combined_motor[6] * motor_norm / (motor_norm * motor_norm);
//#pragma gpc multivector_component out_motor e0^e1^e2^e3 out_motor[7]
out_motor[7] = combined_motor[7] * motor_norm / (motor_norm * motor_norm);

#line 164 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"


calculatedMotor[0] = out_motor[0];
calculatedMotor[1] = out_motor[1];
calculatedMotor[2] = out_motor[2];
calculatedMotor[3] = out_motor[3];
calculatedMotor[4] = out_motor[4];
calculatedMotor[5] = out_motor[5];
calculatedMotor[6] = out_motor[6];
calculatedMotor[7] = out_motor[7];



#line 169 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"

    // Set the calculated motor to the out motors
    for(int i=0; i<8; i++) {
        outMotors[8*idx + i] = calculatedMotor[i];
    }
}

/*
 * Sum up all motors in parallel. Implementation based on https://sodocumentation.net/cuda/topic/6566/parallel-reduction--e-g--how-to-sum-an-array-
 */
__global__ void sumMotors(const float *calculatedMotors, float* summedMotor, int motorsCount, float stepSize) {
    // 2D Thread ID
    int idx = threadIdx.x;

    // Sum up locally first, if there are more motors than the block size
    float localSum[8] = {0};
    for (int i = idx; i < motorsCount; i += blockDim.x) {
        #pragma unroll
        for(int j=0; j<8; j++) {
            localSum[j] += calculatedMotors[8*i + j];
        }
    }

    // Initialize a shared array for summing
    extern __shared__ float partialSum[];

    // Apply the current value if it is not nan
    if(!isnan(localSum[0])) {
        #pragma unroll
        for(int i=0; i<8; i++) {
            partialSum[8*idx+i] = localSum[i];
        }
    }

    // Wait until all local calculations are done
    __syncthreads();

    // Do the graph based sum (adapted from the PMPP lecture slides "2021-10-26-CUDAProgramming3" slide 55)
    unsigned int t = threadIdx.x;
    unsigned int stride;
    for (stride = blockDim.x; stride > 1;) {
        __syncthreads();
        stride = stride >> 1;
        if (t < stride) {
            #pragma unroll
            for(int j=0; j<8; j++) {
                partialSum[8*t+j] += partialSum[8*(t + stride)+j];
            }
        }
    }


    // Normalize & Scale the summed motor and then join it with the current combined motor (stored in constant memory)
    __syncthreads();

    // Normalize and scale with step size
    if(idx > 0 && idx < 8) {
        partialSum[idx] = stepSize*(partialSum[idx] / partialSum[0]);
    }

    __syncthreads();

    if(idx == 0) {
        // Print if wished
        //printf("Sum: [%f,%f,%f,%f,%f,%f,%f,%f]\n", partialSum[0], partialSum[1], partialSum[2], partialSum[3], partialSum[4], partialSum[5], partialSum[6], partialSum[7]);

        // Set the scalar component to 1
        partialSum[0] = 1;

        // Join with input motor
        // Init output
        float joinedMotor[8] = {0.0};



#line 248 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"
#include <math.h>
//#pragma gpc multivector combined
float combined[8];
//#pragma gpc multivector combined_norm
float combined_norm;
//#pragma gpc multivector normed_motor
float normed_motor[8];

//#pragma gpc multivector_component combined 1.0 combined[0]
combined[0] = partialSum[0] * constantMotor[0] + (-(partialSum[4] * constantMotor[4])) + (-(partialSum[5] * constantMotor[5])) + (-(partialSum[6] * constantMotor[6]));
//#pragma gpc multivector_component combined e0^e1 combined[1]
combined[1] = partialSum[0] * constantMotor[1] + partialSum[1] * constantMotor[0] + (-(partialSum[2] * constantMotor[4])) + (-(partialSum[3] * constantMotor[5])) + partialSum[4] * constantMotor[2] + partialSum[5] * constantMotor[3] + (-(partialSum[6] * constantMotor[7])) + (-(partialSum[7] * constantMotor[6]));
//#pragma gpc multivector_component combined e0^e2 combined[2]
combined[2] = partialSum[0] * constantMotor[2] + partialSum[1] * constantMotor[4] + partialSum[2] * constantMotor[0] + (-(partialSum[3] * constantMotor[6])) + (-(partialSum[4] * constantMotor[1])) + partialSum[5] * constantMotor[7] + partialSum[6] * constantMotor[3] + partialSum[7] * constantMotor[5];
//#pragma gpc multivector_component combined e0^e3 combined[3]
combined[3] = partialSum[0] * constantMotor[3] + partialSum[1] * constantMotor[5] + partialSum[2] * constantMotor[6] + partialSum[3] * constantMotor[0] + (-(partialSum[4] * constantMotor[7])) + (-(partialSum[5] * constantMotor[1])) + (-(partialSum[6] * constantMotor[2])) + (-(partialSum[7] * constantMotor[4]));
//#pragma gpc multivector_component combined e1^e2 combined[4]
combined[4] = partialSum[0] * constantMotor[4] + partialSum[4] * constantMotor[0] + (-(partialSum[5] * constantMotor[6])) + partialSum[6] * constantMotor[5];
//#pragma gpc multivector_component combined e1^e3 combined[5]
combined[5] = partialSum[0] * constantMotor[5] + partialSum[4] * constantMotor[6] + partialSum[5] * constantMotor[0] + (-(partialSum[6] * constantMotor[4]));
//#pragma gpc multivector_component combined e2^e3 combined[6]
combined[6] = partialSum[0] * constantMotor[6] + (-(partialSum[4] * constantMotor[5])) + partialSum[5] * constantMotor[4] + partialSum[6] * constantMotor[0];
//#pragma gpc multivector_component combined e0^e1^e2^e3 combined[7]
combined[7] = partialSum[0] * constantMotor[7] + partialSum[1] * constantMotor[6] + (-(partialSum[2] * constantMotor[5])) + partialSum[3] * constantMotor[4] + partialSum[4] * constantMotor[3] + (-(partialSum[5] * constantMotor[2])) + partialSum[6] * constantMotor[1] + partialSum[7] * constantMotor[0];
//#pragma gpc multivector_component combined_norm 1.0 combined_norm
combined_norm = sqrtf(fabs(combined[0] * combined[0] + (-(combined[4] * (-combined[4]))) + (-(combined[5] * (-combined[5]))) + (-(combined[6] * (-combined[6])))));
//#pragma gpc multivector_component normed_motor 1.0 normed_motor[0]
normed_motor[0] = combined[0] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e0^e1 normed_motor[1]
normed_motor[1] = combined[1] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e0^e2 normed_motor[2]
normed_motor[2] = combined[2] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e0^e3 normed_motor[3]
normed_motor[3] = combined[3] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e1^e2 normed_motor[4]
normed_motor[4] = combined[4] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e1^e3 normed_motor[5]
normed_motor[5] = combined[5] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e2^e3 normed_motor[6]
normed_motor[6] = combined[6] * combined_norm / (combined_norm * combined_norm);
//#pragma gpc multivector_component normed_motor e0^e1^e2^e3 normed_motor[7]
normed_motor[7] = combined[7] * combined_norm / (combined_norm * combined_norm);

#line 257 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"


joinedMotor[0] = normed_motor[0];
joinedMotor[1] = normed_motor[1];
joinedMotor[2] = normed_motor[2];
joinedMotor[3] = normed_motor[3];
joinedMotor[4] = normed_motor[4];
joinedMotor[5] = normed_motor[5];
joinedMotor[6] = normed_motor[6];
joinedMotor[7] = normed_motor[7];


#line 261 "D:/Development/GAAlign/src/optimization/gradient_descent_cuda/gradient_descent_cuda.cug"

        //printf("joinedMotor: [%f,%f,%f,%f,%f,%f,%f,%f]\n", joinedMotor[0], joinedMotor[1], joinedMotor[2], joinedMotor[3], joinedMotor[4], joinedMotor[5], joinedMotor[6], joinedMotor[7]);

        // Apply to output -> this is the slowest part!
        #pragma unroll
        for(int i=0; i<8; i++) {
            summedMotor[i] = joinedMotor[i];
        }
    }

}

gaalign::Motor gaalign::GradientDescentOptimizerCUDA::optimize(const std::vector<Correspondence> &correspondences) const {
    // Sanity checks
    if(!m_initialized) {
        std::cout << "ERROR: GradientDescentOptimizerCUDA needs to be initialized using init() before use!";
    }

    // If there are more correspondences than we need -> Randomly sample the chosen amount
    std::vector<Correspondence> sampledCorrespondences;
    sampledCorrespondences.reserve(m_settings.maxCorrespondences);
    if(correspondences.size() > m_settings.maxCorrespondences) {
        // Initialize an array containing the indices of all correspondences
        std::vector<std::uint32_t> indices(correspondences.size());
        std::iota(indices.begin(), indices.end(), 0); // Fill with values [0, 1, 2, ..]

        // Shuffle the array
        shuffle_pcg_divisionless_with_slight_bias(indices.data(), indices.size());

        // Use the first n indices
        for(int i=0; i<m_settings.maxCorrespondences; i++) {
            sampledCorrespondences.push_back(correspondences[indices[i]]);
        }
    }
    else {
        sampledCorrespondences = correspondences;
    }

    if(m_settings.verbose) std::cout << "Running CUDA Gradient descent.." << std::endl;
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    // Put the correspondences into a large array
    #pragma omp parallel for
    for (int i = 0; i < sampledCorrespondences.size(); i++) {
        correspondenceArrayCPU[6 * i] =     (float)sampledCorrespondences[i].first.x();
        correspondenceArrayCPU[6 * i + 1] = (float)sampledCorrespondences[i].first.y();
        correspondenceArrayCPU[6 * i + 2] = (float)sampledCorrespondences[i].first.z();
        correspondenceArrayCPU[6 * i + 3] = (float)sampledCorrespondences[i].second.x();
        correspondenceArrayCPU[6 * i + 4] = (float)sampledCorrespondences[i].second.y();
        correspondenceArrayCPU[6 * i + 5] = (float)sampledCorrespondences[i].second.z();
    }

    // Upload the correspondences to the device
    hipMemcpy(correspondencesGPU, (float*)correspondenceArrayCPU, 6 * sizeof(float) * sampledCorrespondences.size(), hipMemcpyHostToDevice); CUDA_CHECK_ERROR;

    // Sync the devices
    hipDeviceSynchronize();

    // Measure only upload
    auto endUpload = std::chrono::high_resolution_clock::now();
    double timeMSUpload = ((double)std::chrono::duration_cast<std::chrono::nanoseconds>(endUpload - begin).count())/1000000.0;

    // Upload an identity motor to the current constant motor
    gaalign::Motor identity = gaalign::Motor::identity();
    float* identityMotorFloat = (float*) malloc(8 * sizeof(float));
    for(int i=0; i<8; i++) {
        identityMotorFloat[i] = (float)identity.data[i];
    }
    hipMemcpyToSymbol(HIP_SYMBOL(constantMotor), (float*)identityMotorFloat, 8 * sizeof(float), 0, hipMemcpyHostToDevice); CUDA_CHECK_ERROR;

    // Do the actual iterations
    for (int iter = 0; iter < m_settings.maxIterations; iter++) {
        // Execute the main kernel that computes the motors
        calculateMotors<<<1, 512, 6*sampledCorrespondences.size()*sizeof(float)>>>(correspondencesGPU, indicesGPU, calculatedMotors, iter, m_settings.trianglesPerIteration, sampledCorrespondences.size()); CUDA_CHECK_ERROR;

        // Execute a reduction kernel that calculates the averaged motor
        int reductionBlockSize = 64; // MUST be a power of two and smaller than the number of motors
        double stepSize = m_settings.stepSize;
        if(iter == 0) stepSize = 1;
        sumMotors<<<1, reductionBlockSize, /*shared memory size*/ 8*reductionBlockSize*sizeof(unsigned int)>>>(calculatedMotors, avgMotor, m_settings.trianglesPerIteration, stepSize); CUDA_CHECK_ERROR;

        // Update the motor in constant memory to the current combined motor
        hipMemcpyToSymbol(HIP_SYMBOL(constantMotor), (float*)avgMotor, 8 * sizeof(float), 0, hipMemcpyDeviceToDevice); CUDA_CHECK_ERROR;
    }

    // Download the final motor and convert it to a Motor struct
    float* resultMotorArr = (float*)malloc(8 * sizeof(float));
    hipMemcpy(resultMotorArr, (float*)avgMotor, 8 * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK_ERROR;

    // Unpack to motor
    Motor result;
    for(int i=0; i<8; i++) {
        result.data[i] = (double)resultMotorArr[i];
    }

    result.print();

    // End timer
    auto end = std::chrono::high_resolution_clock::now();
    double timeMS = ((double)std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count())/1000000.0;
    if(m_settings.printTiming) std::cout << "Finished Optimization in " << timeMS << " ms (Upload took " << timeMSUpload << " ms)" << std::endl;



    return result;
}

std::string gaalign::GradientDescentOptimizerCUDA::getName() const {
    return "Gradient Descent CUDA";
}

gaalign::GradientDescentSettingsCUDA &gaalign::GradientDescentOptimizerCUDA::getSettings() {
    return m_settings;
}

void gaalign::GradientDescentOptimizerCUDA::init() {
    if(m_initialized) {
        return;
    }

    std::cout << "Initializing memory for the cuda optimizer.." << std::endl;
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    // Allocate memory for correspondences
    hipMalloc(&correspondencesGPU, 6 * sizeof(float) * m_settings.maxCorrespondences);

    // Allocate the memory for indices
    hipMalloc(&indicesGPU, sizeof(unsigned int) * m_settings.trianglesPerIteration*m_settings.maxIterations);

    // Allocate the memory for the motors calculated per iteration
    hipMalloc(&calculatedMotors, sizeof(float) * 8 * m_settings.trianglesPerIteration);

    // Allocate the memory for the averaged motor generated by the kernel
    hipMalloc(&avgMotor, sizeof(float) * 8);

    // Also initialize the CPU memory
    correspondenceArrayCPU = (float *) malloc(6 * sizeof(float) * m_settings.maxCorrespondences);

    // End timer
    auto end = std::chrono::high_resolution_clock::now();
    double timeMS = ((double)std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count())/1000000.0;
    if(m_settings.printTiming) std::cout << "Allocated memory in " << timeMS << " ms" << std::endl;

    // Store that this optimizer was initialized
    m_initialized = true;
}

gaalign::GradientDescentOptimizerCUDA::~GradientDescentOptimizerCUDA() {
    // Free gpu memory
    hipFree(correspondencesGPU);
    hipFree(indicesGPU);
    hipFree(calculatedMotors);
    hipFree(avgMotor);

    // Also free CPU memory
    free(correspondenceArrayCPU);
}
